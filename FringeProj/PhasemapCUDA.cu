#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

///////////////////////////////////////////////////////////////////////////////
#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#endif

#include "PhasemapCUDADll.h"

#pragma unroll

__device__ __constant__ unsigned short ui_d_Bitshift[MAXGRAYBITS];
__device__ unsigned short bps2cilut[BPSLUTSIZE];

//--------------------------------------------------------------------------------------------------
__global__
void CalcBPS2CILutCUDA(unsigned char maxBits, unsigned short *d_tmpBps2cilut)
{
//    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    int b, bitmask = 0, invert = 0;

    b = y;
    for (int i = maxBits; i >= 0; i-- )
    {
        bitmask = 1<<i;
        if ( invert )
            b = b ^ bitmask;
        if ( y & bitmask )
            invert = !invert;
    }
//    d_tmpBps2cilut[y] = b;
    bps2cilut[y] = b;

    return;
}

extern "C" int CalcBPS2CILut(unsigned char numBits)
{
    dim3 dBlock, dGrid;
    unsigned short numThreads=1<<numBits;
    hipError_t cerror;
    unsigned short *d_tempBps2cilut=NULL, *h_tempBps2cilut=NULL;

    hipMalloc((void**)&d_tempBps2cilut, BPSLUTSIZE*sizeof(unsigned short));
    h_tempBps2cilut = (unsigned short*)calloc(BPSLUTSIZE, sizeof(unsigned short));
    CalcDimsVec(numThreads, &dBlock, &dGrid);

    CalcBPS2CILutCUDA<<<dimGrid, dimBlocks>>>(numBits, d_tempBps2cilut);

    hipDeviceSynchronize();
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error invoke kernel calcBPSLut\n" << std::endl;
//        MessageBox(NULL, "Error invoke kernel calcBPSLut", "CDUA", MB_ICONEXCLAMATION|MB_OK);
        return cerror;
    }
//    hipMemcpy(h_tempBps2cilut, d_tempBps2cilut, BPSLUTSIZE*sizeof(unsigned short), hipMemcpyDeviceToHost);
//    hipMemcpyToSymbol(HIP_SYMBOL(bps2cilut), h_tempBps2cilut, BPSLUTSIZE*sizeof(unsigned short));
    hipDeviceSynchronize();
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error copying BPSLut\n" << std::endl;
//        MessageBox(NULL, "Error copying BPSLut", "CDUA", MB_ICONEXCLAMATION|MB_OK);
        return cerror;
    }

    hipFree(d_tempBps2cilut);
    free(h_tempBps2cilut);

    return 0;
}

//--------------------------------------------------------------------------------------------------
__global__
void BPS2CIMapGpu(unsigned char maxBits, int pitchCiMap, short *ui_d_CiMap)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
    unsigned int b, g, bitmask = 0;
    unsigned char invert = 0;

    g = b = ((short*)((char*)ui_d_CiMap + y * pitchCiMap))[x];
    for (int i = maxBits; i >= 0; i--)
    {
        bitmask = 1 << i;
        if (invert)
        {
            b = b ^ bitmask;
        }
        if (g & bitmask)
        {
            invert = !invert;
        }
    }
    ((short*)((char*)ui_d_CiMap + y * pitchCiMap))[x] = b;
}


template<typename _Tp> __global__
void calcBPSGpu(float f_contThreas, struct hipPitchedPtr pp_d_images, struct hipExtent extent, int pitchBPS, short *ui_d_BPS)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    char* devPtr = (char*)pp_d_images.ptr;
    int pitch = pp_d_images.pitch;
    int slicePitch = pitch * extent.height;
    float threas;
    unsigned short bitplanestack;

    if ((((_Tp*)(devPtr + slicePitch + y * pitch))[x] - ((_Tp*)(devPtr + y * pitch))[x]) > f_contThreas)
    {
        threas = (((_Tp*)((char *)devPtr + y * pitch))[x] + ((_Tp*)((char *)devPtr + slicePitch + y * pitch))[x]) / 2.0;

        bitplanestack = 0;

        for(int imgNr = 2; imgNr < extent.depth; imgNr++)
        {
            if (((_Tp*)(devPtr + slicePitch * imgNr + y * pitch))[x] > threas)
            {
                bitplanestack |= ui_d_Bitshift[imgNr - 2];
            }
        }

//        ((unsigned short*)((char*)ui_d_BPS+y*pitchBPS))[x] = bitplanestack;
        ((short*)((char*)ui_d_BPS + y * pitchBPS))[x] = bps2cilut[bitplanestack];
    }
    else
    {
        ((short*)((char*)ui_d_BPS + y * pitchBPS))[x] = INVPHA;
    }
}

extern "C" template<typename _Tp> int CalcCIMap(struct tvArray3D **images, float contThreas, struct tvArray2D **CiMap)
{
    dim3 dimBlocks, dimGrid;
    int ret = 0;
    hipError_t cerror;
    struct hipExtent imageExt;
    struct hipPitchedPtr pp_d_images;
    struct hipMemcpy3DParms imgMemcpyParms;
    int pitchCiMap;
    short *ui_d_CiMap = NULL;
    unsigned short bitshift[MAXGRAYBITS];

//    if ((images == NULL) || (CiMap == NULL) || (cudaInit == 0))
//    {
//        return EPARAM;
//    }

//    if ((*images == NULL) || ((*images)->sizes[1] != (*CiMap)->sizes[0]) || ((*images)->sizes[2] != (*CiMap)->sizes[1]))
//    {
//        return EPARAM;
//    }

    CalcDims((*images)->sizes[2], (*images)->sizes[1], &dimBlocks, &dimGrid);

    imageExt.width = (*images)->sizes[2] * sizeof(_Tp);
    imageExt.height = (*images)->sizes[1];
    imageExt.depth = (*images)->sizes[0];

    hipMalloc3D(&pp_d_images, imageExt);
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error malloc images - CUDA cimap\n" << std::endl;
//        MessageBox(NULL, "Error malloc images", "CDUA - CiMap", MB_ICONEXCLAMATION|MB_OK);
        ret = cerror;
        goto end;
    }

    imgMemcpyParms.srcArray = 0;
    imgMemcpyParms.dstArray = 0;
    imgMemcpyParms.srcPos = make_hipPos(0, 0, 0);
    imgMemcpyParms.dstPtr = pp_d_images;
    imgMemcpyParms.extent = imageExt;
    imgMemcpyParms.kind = hipMemcpyHostToDevice;
    imgMemcpyParms.srcPtr = make_hipPitchedPtr((*images)->vals, (*images)->sizes[2] * sizeof(_Tp), (*images)->sizes[2], (*images)->sizes[1]);
    imgMemcpyParms.dstPos = make_hipPos(0, 0, 0);
    hipMemcpy3D(&imgMemcpyParms);

    hipMallocPitch((void**)&ui_d_CiMap, &pitchCiMap, (*images)->sizes[2] * sizeof(short), (*images)->sizes[1]);

    if ((ret = CalcBPS2CILut(uc_numBits)))
    {
        return ret;
    }

    memset(bitshift, 0, MAXGRAYBITS * sizeof(unsigned short));
    for (int g = 0; g < uc_numBits; g++)
    {
        bitshift[g] = 1 << (uc_numBits - g - 1);
    }

    hipMemcpyToSymbol(HIP_SYMBOL(ui_d_Bitshift), bitshift, MAXGRAYBITS * sizeof(unsigned short), 0, hipMemcpyHostToDevice);
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error copy bitshift vector\n" << std::endl;
//        MessageBox(NULL, "Error copy bitshift vector", "CDUA", MB_ICONEXCLAMATION|MB_OK);
        return cerror;
    }

    calcBPSUCGpu<<<dimGrid, dimBlocks>>>(f_contThreas, pp_d_images, imageExt, pitchCiMap, ui_d_CiMap);
    hipDeviceSynchronize();
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error invoke kernel BPS\n" << std::endl;
//        MessageBox(NULL, "Error invoke kernel BPS", "CDUA", MB_ICONEXCLAMATION|MB_OK);
        return cerror;
    }

    hipMemcpy2D((*CiMap)->vals, (*images)->sizes[2]*sizeof(short), ui_d_CiMap, pitchCiMap, (*images)->sizes[2]*sizeof(short), (*images)->sizes[1], hipMemcpyDeviceToHost);

end:
    if (ui_d_CiMap)
    {
        hipFree(ui_d_CiMap);
    }
    if (pp_d_images.ptr)
    {
        hipFree(pp_d_images.ptr);
    }

    return ret;
}

//--------------------------------------------------------------------------------------------------
template<typename _Tp> __global__
void calcPhaseMap4Gpu(float f_contThreas, _Tp overExp, struct hipPitchedPtr pp_d_images, struct hipExtent extent, int pitchPhaseMap, CFPTYPE *f_d_phaseMap, int pitchModulationMap, CFPTYPE *f_d_modulationMap)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    char* devPtr = (char*)pp_d_images.ptr;
    int pitch = pp_d_images.pitch;
    int slicePitch = pitch * extent.height;
    _Tp max = 0;
    _Tp Int[4];
    CFPTYPE contrast = 0, buf1 = 0, buf2 = 0;

    for (int n = 0; n < 4; n++)
    {
        Int[n] = ((_Tp*)(devPtr + slicePitch * n + y * pitch))[x];
        if (Int[n] > max)
        {
            max = Int[n];
        }
    }

    buf1 = Int[1] - Int[3];
    buf2 = Int[2] - Int[0];
    contrast = sqrt(buf1 * buf1 + buf2 * buf2);

    ((CFPTYPE*)((char*)f_d_modulationMap + pitchModulationMap * y))[x] = contrast;
    if ((contrast>f_contThreas) && (((overExp) && (max < overExp)) || !overExp))
    {
        ((CFPTYPE*)((char*)f_d_phaseMap + pitchPhaseMap * y))[x] = atan2(buf1, buf2);
    }
    else
    {
        ((CFPTYPE*)((char*)f_d_phaseMap + pitchPhaseMap * y))[x] = INVPHA;
    }
}

extern "C" template<typename _Tp> int CalcPhaseMap4(struct tvArray3D **images, float contThreas, _Tp overExp, struct tFloatArray2D **PhaseMap, struct tFloatArray2D **ModulationMap)
{
    dim3 dimBlocks, dimGrid;
    int ret = 0;
    hipError_t cerror;
    struct hipExtent imageExt;
    struct hipPitchedPtr pp_d_images;
    struct hipMemcpy3DParms imgMemcpyParms;
    int pitchPhaseMap, pitchModulationMap;
    CFPTYPE *f_d_PhaseMap = NULL, *f_d_ModulationMap = NULL;

//    if ((images == NULL) || (PhaseMap == NULL) || (cudaInit == 0))
//    {
//        return EPARAM;
//    }

//    if ((*images == NULL) || ((*images)->sizes[0] != 4)
//        || ((*images)->sizes[1] != (*PhaseMap)->sizes[0])
//        || ((*images)->sizes[2] != (*PhaseMap)->sizes[1]))
//    {
//        return EPARAM;
//    }

    CalcDims((*images)->sizes[2], (*images)->sizes[1], &dimBlocks, &dimGrid);

    imageExt.width = (*images)->sizes[2] * sizeof(_Tp);
    imageExt.height = (*images)->sizes[1];
    imageExt.depth = 4;

    hipMalloc3D(&pp_d_images, imageExt);
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error malloc images - CUDA phasemap 4\n" << std::endl;
//        MessageBox(NULL, "Error malloc images", "CDUA - Phamap 4", MB_ICONEXCLAMATION|MB_OK);
        ret = cerror;
        goto end;
    }

    imgMemcpyParms.srcArray = 0;
    imgMemcpyParms.dstArray = 0;
    imgMemcpyParms.srcPos = make_hipPos(0, 0, 0);
    imgMemcpyParms.dstPtr = pp_d_images;
    imgMemcpyParms.extent = imageExt;
    imgMemcpyParms.kind = hipMemcpyHostToDevice;
    imgMemcpyParms.srcPtr = make_hipPitchedPtr((*images)->vals, (*images)->sizes[2] * sizeof(_Tp), (*images)->sizes[2], (*images)->sizes[1]);
    imgMemcpyParms.dstPos = make_hipPos(0, 0, 0);
    hipMemcpy3D(&imgMemcpyParms);

    hipMallocPitch((void**)&f_d_PhaseMap, &pitchPhaseMap, (*images)->sizes[2] * sizeof(CFPTYPE), (*images)->sizes[1]);
    hipMallocPitch((void**)&f_d_ModulationMap, &pitchModulationMap, (*images)->sizes[2] * sizeof(CFPTYPE), (*images)->sizes[1]);

    calcPhaseMap4Gpu<_Tp><<<dimGrid, dimBlocks>>>(f_contThreas, overExp, pp_d_images, imageExt, pitchPhaseMap, f_d_PhaseMap, pitchModulationMap, f_d_ModulationMap);
    hipDeviceSynchronize();
    if ((cerror = hipGetLastError()))
    {
        ret = cerror;
        goto end;
    }

    hipMemcpy2D((*ModulationMap)->vals, (*images)->sizes[2] * sizeof(CFPTYPE), f_d_ModulationMap, pitchModulationMap, (*images)->sizes[2]*sizeof(CFPTYPE), (*images)->sizes[1], hipMemcpyDeviceToHost);
    hipMemcpy2D((*PhaseMap)->vals, (*images)->sizes[2] * sizeof(CFPTYPE), f_d_PhaseMap, pitchPhaseMap, (*images)->sizes[2]*sizeof(CFPTYPE), (*images)->sizes[1], hipMemcpyDeviceToHost);

end:
    if (f_d_PhaseMap)
    {
        hipFree(f_d_PhaseMap);
    }
    if (f_d_ModulationMap)
    {
        hipFree(f_d_ModulationMap);
    }
    if (pp_d_images.ptr)
    {
        hipFree(pp_d_images.ptr);
    }

    return ret;
}

//--------------------------------------------------------------------------------------------------
template<typename _Tp> __global__
void calcPhaseMapNGpu(unsigned char numImages, CFPTYPE *f_d_sines, CFPTYPE *f_d_cosines, float f_contThreas, _Tp overExp, struct hipPitchedPtr pp_d_images, struct hipExtent extent, int pitchPhaseMap, CFPTYPE *f_d_phaseMap, int pitchModulationMap, CFPTYPE *f_d_modulationMap)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    char* devPtr = (char*)pp_d_images.ptr;
    int pitch = pp_d_images.pitch;
    int slicePitch = pitch * extent.height;
    _Tp max = 0;
    _Tp Int[MAXPHASHIFT];
    CFPTYPE contrast = 0, buf1 = 0, buf2 = 0;

    for (int n = 0; n < numImages; n++)
    {
        Int[n] = ((_Tp*)(devPtr + slicePitch * n + y * pitch))[x];
        if (Int[n] > max)
        {
            max = Int[n];
        }
    }

//    ((CFPTYPE*)((char*)f_d_modulationMap+pitchModulationMap*y))[x] = x;
    for (int n = 0; n < numImages; n++)
    {
         buf1 += Int[n] * f_d_sines[n];
         buf2 += Int[n] * f_d_cosines[n];
    }

    contrast = sqrt(buf1 * buf1 + buf2 * buf2);

    ((CFPTYPE*)((char*)f_d_modulationMap + pitchModulationMap * y))[x] = contrast;
    if ((contrast > f_contThreas) && (((overExp) && (max < overExp)) || !overExp))
    {
        ((CFPTYPE*)((char*)f_d_phaseMap + pitchPhaseMap * y))[x] = atan2(buf1, buf2);
    }
    else
    {
        ((CFPTYPE*)((char*)f_d_phaseMap + pitchPhaseMap * y))[x] = INVPHA;
    }
}

extern "C" template<typename _Tp> int CalcPhaseMapN(struct tvArray3D **images, float contThreas, _Tp overExp, struct tFloatArray2D **PhaseMap, struct tFloatArray2D **ModulationMap)
{
    dim3 dimBlocks, dimGrid;
    int ret = 0;
    hipError_t cerror;
    struct hipExtent imageExt;
    struct hipPitchedPtr pp_d_images;
    struct hipMemcpy3DParms imgMemcpyParms;
    int pitchPhaseMap, pitchModulationMap;
    CFPTYPE *f_d_PhaseMap = NULL, *f_d_ModulationMap = NULL;
    CFPTYPE *f_d_sines = NULL, *f_d_cosines = NULL, *f_h_sines = NULL, *f_h_cosines = NULL;

//    if ((images == NULL) || (PhaseMap == NULL) || (cudaInit == 0))
//    {
//        return EPARAM;
//    }

//    if ((*images == NULL)
//        || ((*images)->sizes[1] != (*PhaseMap)->sizes[0]) || ((*images)->sizes[2] != (*PhaseMap)->sizes[1]))
//    {
//        return EPARAM;
//    }

    CalcDims((*images)->sizes[2], (*images)->sizes[1], &dimBlocks, &dimGrid);

    imageExt.width = (*images)->sizes[2] * sizeof(_Tp);
    imageExt.height = (*images)->sizes[1];
    imageExt.depth = (*images)->sizes[0];

    hipMalloc3D(&pp_d_images, imageExt);
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error malloc images - CUDA phasemap N\n" << std::endl;
//        MessageBox(NULL, "Error malloc images", "CDUA - Phamap N", MB_ICONEXCLAMATION|MB_OK);
        ret = cerror;
        goto end;
    }

    imgMemcpyParms.srcArray = 0;
    imgMemcpyParms.dstArray = 0;
    imgMemcpyParms.srcPos = make_hipPos(0, 0, 0);
    imgMemcpyParms.dstPtr = pp_d_images;
    imgMemcpyParms.extent = imageExt;
    imgMemcpyParms.kind = hipMemcpyHostToDevice;
    imgMemcpyParms.srcPtr = make_hipPitchedPtr((*images)->vals, (*images)->sizes[2] * sizeof(_Tp), (*images)->sizes[2], (*images)->sizes[1]);
    imgMemcpyParms.dstPos = make_hipPos(0, 0, 0);
    hipMemcpy3D(&imgMemcpyParms);

    hipMallocPitch((void**)&f_d_PhaseMap, &pitchPhaseMap, (*images)->sizes[2] * sizeof(CFPTYPE), (*images)->sizes[1]);
    hipMallocPitch((void**)&f_d_ModulationMap, &pitchModulationMap, (*images)->sizes[2] * sizeof(CFPTYPE), (*images)->sizes[1]);

    f_h_sines = (CFPTYPE*)malloc(numImages * sizeof(CFPTYPE));
    f_h_cosines = (CFPTYPE*)malloc(numImages * sizeof(CFPTYPE));
    hipMalloc(&f_d_sines, numImages * sizeof(CFPTYPE));
    hipMalloc(&f_d_cosines, numImages * sizeof(CFPTYPE));

    for (int nimg = 0; nimg < numImages; nimg++)
    {
        f_h_sines[nimg] = sin(nimg * CUDA2PI / (CFPTYPE)(numImages));
        f_h_cosines[nimg] = -1.0 * cos(nimg * CUDA2PI / (CFPTYPE)(numImages));
    }

    hipMemcpy(f_d_sines, f_h_sines, numImages * sizeof(CFPTYPE), hipMemcpyHostToDevice);
    hipMemcpy(f_d_cosines, f_h_cosines, numImages * sizeof(CFPTYPE), hipMemcpyHostToDevice);
    free(f_h_sines);
    free(f_h_cosines);

    calcPhaseMapNGpu<_Tp><<<dimGrid, dimBlocks>>>(numImages, f_d_sines, f_d_cosines, f_contThreas, overExp, pp_d_images, imageExt, pitchPhaseMap, f_d_PhaseMap, pitchModulationMap, f_d_ModulationMap);
    hipDeviceSynchronize();

    hipFree(f_d_cosines);
    hipFree(f_d_sines);

    hipMemcpy2D((*ModulationMap)->vals, (*images)->sizes[2] * sizeof(CFPTYPE), f_d_ModulationMap, pitchModulationMap, (*images)->sizes[2]*sizeof(CFPTYPE), (*images)->sizes[1], hipMemcpyDeviceToHost);
    hipMemcpy2D((*PhaseMap)->vals, (*images)->sizes[2] * sizeof(CFPTYPE), f_d_PhaseMap, pitchPhaseMap, (*images)->sizes[2]*sizeof(CFPTYPE), (*images)->sizes[1], hipMemcpyDeviceToHost);

    if (f_d_PhaseMap)
    {
        hipFree(f_d_PhaseMap);
    }
    if (f_d_ModulationMap)
    {
        hipFree(f_d_ModulationMap);
    }
    if (pp_d_images.ptr)
    {
        hipFree(pp_d_images.ptr);
    }

    return ret;
}

//--------------------------------------------------------------------------------------------------
__global__
void unwrapPhaseGrayGpu(float contThreas, unsigned short maxpha, int pitchCiMap, short *ui_d_CiMap, int pitchRawPhase, CFPTYPE *f_d_RawPhase, int pitchModulationMap, CFPTYPE *f_d_ModulationMap, int pitchPhaseMap, CFPTYPE *f_d_PhaseMap)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    short CiMap = ((short*)((char*)ui_d_CiMap + y * pitchCiMap))[x];
    CFPTYPE rawPhase = ((CFPTYPE*)((char*)f_d_RawPhase + y * pitchRawPhase))[x];

    //Phase Unwrapping mit Codeindizes
    //-Pi/2                    //pi/2
    if((rawPhase >= -CUDAPI2) && (rawPhase <= CUDAPI2))
    {
        if (((CFPTYPE*)((char*)f_d_ModulationMap + y * pitchModulationMap))[x] > contThreas)
        {
            ((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] = rawPhase + CUDAPI + (CiMap / 2) * CUDA2PI;
        }
        else
        {
            ((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] = INVPHA;
        }
    }
    else if(rawPhase > CUDAPI2)
    {
        if (((CFPTYPE*)((char*)f_d_ModulationMap + y * pitchModulationMap))[x] > contThreas)
        {
            ((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] = rawPhase + CUDAPI + ((CiMap + 1) / 2 - 1) * CUDA2PI;
        }
        else
        {
            ((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] = INVPHA;
        }
    }
    else //(rawPhase < CUDAPI2)
    {
        if (((CFPTYPE*)((char*)f_d_ModulationMap + y * pitchModulationMap))[x] > contThreas)
        {
            ((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] = rawPhase + CUDAPI + ((CiMap + 1) / 2) * CUDA2PI;
        }
        else
        {
            ((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] = INVPHA;
        }
    }

    if ((((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] < 0) || (((CFPTYPE*)((char*)f_d_PhaseMap + y * pitchPhaseMap))[x] > maxpha))
    {
        ((CFPTYPE*)((char*)f_d_PhaseMap+y*pitchPhaseMap))[x] = INVPHA;
    }
}

extern "C" int UnwrapPhaseGray(float contThreas, unsigned short maxpha, struct tShortArray2D **CiMap, struct tFloatArray2D **RawPhase, struct tFloatArray2D **ModulationMap, struct tFloatArray2D **PhaseMap)
{
    dim3 dimBlocks, dimGrid;
    hipError_t cerror;
    int ret = 0;
    int pitchRawPhase, pitchPhaseMap, pitchModulationMap, pitchCiMap;
    CFPTYPE *f_d_RawPhase = NULL, *f_d_PhaseMap = NULL, *f_d_ModulationMap = NULL;
    short *ui_d_CiMap = NULL;

//    if ((RawPhase == NULL) || (CiMap == NULL) || (PhaseMap == NULL) || (cudaInit == 0))
//    {
//        return EPARAM;
//    }

//    if (((*CiMap)->sizes[0] != (*PhaseMap)->sizes[0]) || ((*CiMap)->sizes[1] != (*PhaseMap)->sizes[1])
//            || ((*RawPhase)->sizes[0] != (*PhaseMap)->sizes[0])
//            || ((*RawPhase)->sizes[1] != (*PhaseMap)->sizes[1]))
//    {
//        return EPARAM;
//    }

    CalcDims((*CiMap)->sizes[1], (*CiMap)->sizes[0], &dimBlocks, &dimGrid);

    hipMallocPitch((void**)&f_d_RawPhase, &pitchRawPhase, (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[0]);
    hipMallocPitch((void**)&ui_d_CiMap, &pitchCiMap, (*CiMap)->sizes[1]*sizeof(short), (*CiMap)->sizes[0]);
    hipMallocPitch((void**)&f_d_ModulationMap, &pitchModulationMap, (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[0]);
    if ((cerror = hipGetLastError()))
    {
        std::cerr << "Error malloc images - CUDA unwrap\n" << std::endl;
//        MessageBox(NULL, "Error malloc images", "CDUA - Unwrap", MB_ICONEXCLAMATION|MB_OK);
        ret = cerror;
        goto end;
    }

    hipMemcpy2D(f_d_RawPhase, pitchRawPhase, (*RawPhase)->vals, (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[0], hipMemcpyHostToDevice);
    hipMemcpy2D(ui_d_CiMap, pitchCiMap, (*CiMap)->vals, (*CiMap)->sizes[1]*sizeof(short), (*CiMap)->sizes[1]*sizeof(short), (*CiMap)->sizes[0], hipMemcpyHostToDevice);
    hipMemcpy2D(f_d_ModulationMap, pitchModulationMap, (*ModulationMap)->vals, (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[0], hipMemcpyHostToDevice);

    hipMallocPitch((void**)&f_d_PhaseMap, &pitchPhaseMap, (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[0]);

    unwrapPhaseGrayGpu<<<dimGrid, dimBlocks>>>(contThreas, maxpha, pitchCiMap, ui_d_CiMap, pitchRawPhase, f_d_RawPhase, pitchModulationMap, f_d_ModulationMap, pitchPhaseMap, f_d_PhaseMap);
    hipDeviceSynchronize();
    if ((cerror = hipGetLastError()))
    {
        ret = cerror;
        goto end;
    }

    hipMemcpy2D((*PhaseMap)->vals, (*CiMap)->sizes[1]*sizeof(CFPTYPE), f_d_PhaseMap, pitchPhaseMap, (*CiMap)->sizes[1]*sizeof(CFPTYPE), (*CiMap)->sizes[0], hipMemcpyDeviceToHost);

end:
    if (f_d_RawPhase)
    {
        hipFree(f_d_RawPhase);
    }
    if (ui_d_CiMap)
    {
        hipFree(ui_d_CiMap);
    }
    if (f_d_ModulationMap)
    {
        hipFree(f_d_ModulationMap);
    }
    if (f_d_PhaseMap)
    {
        hipFree(f_d_PhaseMap);
    }

    return 0;
}

//--------------------------------------------------------------------------------------------------
//extern "C" int CLoadLUT(unsigned short *ui_h_BPS2CITable)
//{
//    hipError_t cerror;

///*
//    hipMemcpyToSymbol(HIP_SYMBOL(ui_d_BPS2CITable), ui_h_BPS2CITable, (1<<(MAXGRAYBITS+1))*sizeof(unsigned short), 0, hipMemcpyHostToDevice);
//    if ((cerror = hipGetLastError()))
//    {
//        return cerror;
//    }
//*/
//    return 0;
//}

//--------------------------------------------------------------------------------------------------
extern "C" int InitCudaDevice(int num)
{
    struct hipDeviceProp_t prop;
    int numdev, cudadev;
    hipError_t cerror;

    hipDeviceReset();
    cerror = hipGetLastError();
    cerror = hipGetDeviceCount(&numdev);
    if (numdev == 0)
    {
        return -1;
    }

//    cerror = hipSetDevice(num);
//    cudadev = num;

    for (int r = 0; r < numdev; r++)
    {
        cerror = hipGetDeviceProperties(&prop, r);
        if (prop.major >= 1)
        {
            cerror = hipSetDevice(r);
            cudadev = r;
            break;
        }

        if (r == numdev - 1)
        {
            std::cerr << "No CUDA capable device found\n" << std::endl;
//          cout << "No CUDA capable device found!\nAborting!\n";
            return -1;
        }
    }
//    hipDeviceReset();

    return 0;
}

//--------------------------------------------------------------------------------------------------
extern "C" int CalcDimsVec(long length, dim3 *dimBlock, dim3 *dimGrid)
{
    unsigned short maxbsize = 16;
    unsigned short MaxBlocks = 256;
    int devnum = -1;
    struct hipDeviceProp_t prop;
    hipError_t cerror;

//    hipGetDevice(&devnum);
//    cerror = hipGetDeviceProperties(&prop, devnum);
//    MaxBlocks = prop.maxThreadsPerBlock;
//    maxbsize = (double)MaxBlocks;
    (*dimBlock).y = MaxBlocks;
/*
        for (int n = maxbsize; n > 0; n--)
    {
            if ((floor((float)length / (float)n) == ((float)length / (float)n)) && (n))
            {
                (*dimBlock).x = n;
                break;
            }
    }
*/
    (*dimBlock).x = 1;

    (*dimGrid).y = (length + (*dimBlock).y - 1) / (*dimBlock).y;
        //> make dims a multiple of 16 for faster calculation (see CUDA doku)
        (*dimGrid).y = ceil((*dimGrid).y / 16.0) * 16;
    (*dimGrid).x = 1;
/*
    char buf[200];
    sprintf(buf, "Len: %d\nMaxB: %d\nBlocks: %d\nGrid: %d", length, maxbsize, (*dimBlock).x, (*dimGrid).x);
    MessageBox(NULL, buf, "", MB_OK);
*/
    return 0;
}

//--------------------------------------------------------------------------------------------------
extern "C" int CalcDims(long width, long height, dim3 *dimBlock, dim3 *dimGrid)
{
    unsigned short maxbsize = 16;
    unsigned short MaxBlocks = 128;
    int devnum;
    struct hipDeviceProp_t prop;
    hipError_t cerror;

//    hipGetDevice(&devnum);
//    cerror = hipGetDeviceProperties(&prop, devnum);
//    MaxBlocks = prop.maxThreadsPerBlock;
//    maxbsize = floor(sqrt((double)MaxBlocks));

    for (int n = maxbsize; n > 0; n--)
    {
        if ((floor((CFPTYPE)width / (CFPTYPE)n) == ((CFPTYPE)width / (CFPTYPE)n)) && (n))
        {
            (*dimBlock).x = n;
            break;
        }
    }
    (*dimBlock).x = ceil((*dimBlock).x / 16.0) * 16;
    maxbsize = floor((CFPTYPE)MaxBlocks / (CFPTYPE)(*dimBlock).x);
    for (int n = maxbsize; n > 0; n--)
    {
        if (floor((CFPTYPE)height / (CFPTYPE)n) == ((CFPTYPE)height / (CFPTYPE)n))
        {
            (*dimBlock).y = n;
            break;
        }
    }
    (*dimBlock).y = ceil((*dimBlock).y / 16.0) * 16;
    (*dimGrid).x = (width + (*dimBlock).x - 1) / (*dimBlock).x;
    (*dimGrid).y = (height + (*dimBlock).y - 1) / (*dimBlock).y;

    return 0;
}

//--------------------------------------------------------------------------------------------------
