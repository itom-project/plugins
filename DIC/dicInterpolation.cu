#include "hip/hip_runtime.h"
/* ********************************************************************
    Plugin "DIC" for itom software
    URL: http://lccv.ufal.br/
    Copyright (C) 2016, Universidade Federal de Alagoas (UFAL), Brazil

    This file is part of a plugin for the measurement software itom.

    This itom-plugin is free software; you can redistribute it and/or modify it
    under the terms of the GNU Library General Public Licence as published by
    the Free Software Foundation; either version 2 of the Licence, or (at
    your option) any later version.

    itom and its plugins are distributed in the hope that it will be useful, but
    WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU Library
    General Public Licence for more details.

    You should have received a copy of the GNU Library General Public License
    along with itom. If not, see <http://www.gnu.org/licenses/>.
*********************************************************************** */

#include <vector>
#include "dicInterpolation.hu"
//#include "dicInterpolationMats.h"
#include "dicInterpolationBMatBiCu.h"

#include "opencv2/imgproc/imgproc.hpp"

struct dImgMat {
    int sizex;
    int sizey;
    int step;
    void *d_ptr;
    void *h_ptr;

    dImgMat() : sizex(0), sizey(0), step(0), d_ptr(NULL), h_ptr(NULL) {}
};

std::vector<struct dImgMat> dImgBuffers;

//extern double BMatBiCu[256];
//extern double BMatBiQu[1296];
//extern double BMatBiQi[4096];
// Matrices are stored in row-major order:
// M(row, col) = *(M.elements + row * M.stride + col)
/*
template<typename _Tp> class Matrix {
    int width;
    int height;
    size_t step;
    char colWise;
    _Tp* elements;

    Matrix() : width(0), height(0), step(0), colWise(0), elements(NULL) {}
};

template<typename _Tp> class Vector {
    int length;
    _Tp *elements;

    Vector() : length(0), elements(NULL) {}
};
*/
//--------------------------------------------------------------------------------------------------
/*
// Get a matrix element
template<typename _Tp> __device__ float d_GetElement(const Matrix<_Tp> &A, int row, int col)
{
    if (A.colWise)
        return A.elements[row * A.stride + col];
    else
        return A.elements[col * A.stride + row];
}

//--------------------------------------------------------------------------------------------------
// Set a matrix element
template<typename _Tp> __device__ void d_SetElement(Matrix<_Tp> A, int row, int col,
                           _Tp value)
{
    if (A.colWise)
        A.elements[row * A.stride + col] = value;
    else
        A.elements[col * A.stride + row] = value;
}

//--------------------------------------------------------------------------------------------------
// Get the BLOCK_SIZExBLOCK_SIZE sub-matrix Asub of A that is
// located col sub-matrices to the right and row sub-matrices down
// from the upper-left corner of A
 template<typename _Tp> __device__ void d_GetSubMatrix(Matrix<_Tp> &A, int row, int col)
{
    Matrix<_Tp> Asub;
    Asub.width    = BLOCK_SIZE;
    Asub.height   = BLOCK_SIZE;
    Asub.stride   = A.stride;
    if (A.colPri)
        Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row
                                            + BLOCK_SIZE * col];
    else
        Asub.elements = &A.elements[A.stride * BLOCK_SIZE * col
                                            + BLOCK_SIZE * row];
    return;
}
*/
//--------------------------------------------------------------------------------------------------
__global__ void d_interpolBiCu(const float* __restrict__ pts, int numPts,
    const float* __restrict__ imgIn, int width, int height, int step, int colWise, const float* __restrict__ dxp,
    const float* __restrict__ dyp, const float* __restrict__ dxyp, float* __restrict__ intensVals)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int ptId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (ptId < numPts)
    {
        float fIdxPos;
        float sIdxPos;
        int sIdx1, fIdx1;
        int fSize, sSize;

        if (colWise)
        {
            fIdxPos = pts[ptId * 2 + 1];
            sIdxPos = pts[ptId * 2];
            fSize = height;
            sSize = width;
        }
        else
        {
            fIdxPos = pts[ptId * 2];
            sIdxPos = pts[ptId * 2 + 1];
            fSize = width;
            sSize = height;
        }

        if ((fIdxPos >= fSize) || (sIdxPos >= sSize) || (fIdxPos < 0) || (sIdxPos < 0)
            || !isfinite(fIdxPos) || !isfinite(sIdxPos))
            return;

        if (sIdxPos < (sSize - 1))
        {
            sIdx1 = (int)floor(sIdxPos);
        }
        else
        {
            sIdx1 = (int)floor(sIdxPos - 1);
        }
        if (fIdxPos < (fSize - 1))
        {
            fIdx1 = (int)floor(fIdxPos);
        }
        else
        {
            fIdx1 = (int)floor(fIdxPos - 1);
        }

        float h = sIdxPos - sIdx1;
        float t = fIdxPos - fIdx1;
        float t2 = t * t, t3 = t * t * t;
        float h2 = h * h, h3 = h * h * h;

        // calculate interpolated intensity by multiplying ht with A
        // ht = [1     t        t^2        t^3 ...
        //       h     h * t    h*t^2      h * t^3 ...
        //       h^2   h^2 * t  h^2 * t^2  h^2 * t^3 ...
        //       h^3   h^3 * t  h^3 * t^2  h^3 * t^3];

        // wikipedia version
        float coeffVec[16];
        float I0 = imgIn[sIdx1 * step + fIdx1];
        float I1 = imgIn[sIdx1 * step + fIdx1 + 1];
        float I2 = imgIn[(sIdx1 + 1) * step + fIdx1];
        float I3 = imgIn[(sIdx1 + 1) * step + fIdx1 + 1];
        float DX0 = dxp[sIdx1 * step + fIdx1];
        float DX1 = dxp[sIdx1 * step + fIdx1 + 1];
        float DX2 = dxp[(sIdx1 + 1) * step + fIdx1];
        float DX3 = dxp[(sIdx1 + 1) * step + fIdx1 + 1];
        float DY0 = dyp[sIdx1 * step + fIdx1];
        float DY1 = dyp[sIdx1 * step + fIdx1 + 1];
        float DY2 = dyp[(sIdx1 + 1) * step + fIdx1];
        float DY3 = dyp[(sIdx1 + 1) * step + fIdx1 + 1];
        float DXY0 = dxyp[sIdx1 * step + fIdx1];
        float DXY1 = dxyp[sIdx1 * step + fIdx1 + 1];
        float DXY2 = dxyp[(sIdx1 + 1) * step + fIdx1];
        float DXY3 = dxyp[(sIdx1 + 1) * step + fIdx1 + 1];
        coeffVec[0] = BMatBiCu0    * I0;
        coeffVec[1] = BMatBiCu20    * DX0;
        coeffVec[2] = BMatBiCu32    * I0
            + BMatBiCu33    * I1
            + BMatBiCu36    * DX0
            + BMatBiCu37    * DX1;
        coeffVec[3] = BMatBiCu48    * I0
            + BMatBiCu49    * I1
            + BMatBiCu52    * DX0
            + BMatBiCu53    * DX1;
        coeffVec[4] = BMatBiCu72    * DY0;
        coeffVec[5] = BMatBiCu92    * DXY0;
        coeffVec[6] = BMatBiCu104    * DY0
            + BMatBiCu105    * DY1
            + BMatBiCu108    * DXY0
            + BMatBiCu109    * DXY1;
        coeffVec[7] = BMatBiCu120    * DY0
            + BMatBiCu121    * DY1
            + BMatBiCu124    * DXY0
            + BMatBiCu125    * DXY1;
        coeffVec[8] = BMatBiCu128    * I0
            + BMatBiCu130    * I2
            + BMatBiCu136    * DY0
            + BMatBiCu138    * DY2;
        coeffVec[9] = BMatBiCu148    * DX0
            + BMatBiCu150    * DX2
            + BMatBiCu156    * DXY0
            + BMatBiCu158    * DXY2;
        coeffVec[10] = BMatBiCu160    * I0
            + BMatBiCu161    * I1
            + BMatBiCu162    * I2
            + BMatBiCu163    * I3
            + BMatBiCu164    * DX0
            + BMatBiCu165    * DX1
            + BMatBiCu166    * DX2
            + BMatBiCu167    * DX3
            + BMatBiCu168    * DY0
            + BMatBiCu169    * DY1
            + BMatBiCu170    * DY2
            + BMatBiCu171    * DY3
            + BMatBiCu172    * DXY0
            + BMatBiCu173    * DXY1
            + BMatBiCu174    * DXY2
            + BMatBiCu175    * DXY3;
        coeffVec[11] = BMatBiCu176    * I0
            + BMatBiCu177    * I1
            + BMatBiCu178    * I2
            + BMatBiCu179    * I3
            + BMatBiCu180    * DX0
            + BMatBiCu181    * DX1
            + BMatBiCu182    * DX2
            + BMatBiCu183    * DX3
            + BMatBiCu184    * DY0
            + BMatBiCu185    * DY1
            + BMatBiCu186    * DY2
            + BMatBiCu187    * DY3
            + BMatBiCu188    * DXY0
            + BMatBiCu189    * DXY1
            + BMatBiCu190    * DXY2
            + BMatBiCu191    * DXY3;
        coeffVec[12] = BMatBiCu192    * I0
            + BMatBiCu194    * I2
            + BMatBiCu200    * DY0
            + BMatBiCu202    * DY2;
        coeffVec[13] = BMatBiCu212    * DX0
            + BMatBiCu214    * DX2
            + BMatBiCu220    * DXY0
            + BMatBiCu222    * DXY2;
        coeffVec[14] = BMatBiCu224    * I0
            + BMatBiCu225    * I1
            + BMatBiCu226    * I2
            + BMatBiCu227    * I3
            + BMatBiCu228    * DX0
            + BMatBiCu229    * DX1
            + BMatBiCu230    * DX2
            + BMatBiCu231    * DX3
            + BMatBiCu232    * DY0
            + BMatBiCu233    * DY1
            + BMatBiCu234    * DY2
            + BMatBiCu235    * DY3
            + BMatBiCu236    * DXY0
            + BMatBiCu237    * DXY1
            + BMatBiCu238    * DXY2
            + BMatBiCu239    * DXY3;
        coeffVec[15] = BMatBiCu240    * I0
            + BMatBiCu241    * I1
            + BMatBiCu242    * I2
            + BMatBiCu243    * I3
            + BMatBiCu244    * DX0
            + BMatBiCu245    * DX1
            + BMatBiCu246    * DX2
            + BMatBiCu247    * DX3
            + BMatBiCu248    * DY0
            + BMatBiCu249    * DY1
            + BMatBiCu250    * DY2
            + BMatBiCu251    * DY3
            + BMatBiCu252    * DXY0
            + BMatBiCu253    * DXY1
            + BMatBiCu254    * DXY2
            + BMatBiCu255    * DXY3;

        intensVals[ptId * 3] = coeffVec[0] + coeffVec[1] * t + coeffVec[2] * t2 + coeffVec[3] * t3
            + coeffVec[4] * h + coeffVec[5] * h * t + coeffVec[6] * h * t2 + coeffVec[7] * h * t3
            + coeffVec[8] * h2 + coeffVec[9] * h2 * t + coeffVec[10] * h2 * t2 + coeffVec[11] * h2 * t3
            + coeffVec[12] * h3 + coeffVec[13] * h3 * t + coeffVec[14] * h3 * t2 + coeffVec[15] * h3 * t3;

        //if (flags & 1)
        {
            // calculate interpolated derivative(s) dhdx(t)
            // dhdxt = a(4) + a(5) * t + a(6) * t^2 + a(7) * t^3 ...
            //        + 2 * a(8) * h + 2 * a(9) * h * t + 2 * a(10) * h * t^2 + 2 * a(11) * h * t^3 ...
            //        + 3 * a(12) * h^2 + 3 * a(13) * h^2 * t + 3 * a(14) * h^2 * t^2 + 3 * a(15) * h^2 * t^3;
            intensVals[ptId * 3 + 1] = coeffVec[1] + coeffVec[2] * 2.0 * t + coeffVec[3] * 3.0 * t2 + coeffVec[5] * h
                + coeffVec[6] * 2.0 * h * t + coeffVec[7] * 3.0 * h * t2 + coeffVec[9] * h2 + coeffVec[10] * 2.0 * h2 * t
                + coeffVec[11] * 3.0 * h2 * t2 + coeffVec[13] * h3 + coeffVec[14] * 2.0 * h3 * t + coeffVec[15] * 3.0 * h3 * t2;

            // calculate interpolated derivative(s) dhdy(t)
            // dhdyt = a(1) + 2 * a(2) * t + 3 * a(3) * t^2 + ...
            //    a(5) * h + 2 * a(6) * h * t + 3 * a(7) * h * t^2 + ...
            //    a(9) * h^2 + 2 * a(10) * h^2 * t + 3 * a(11) * h^2 * t^2 + ...
            //    a(13) * h^3 + 2 * a(14)v* h^3 * t + 3 * a(15) * h^3 * t^2;
            intensVals[ptId * 3 + 2] = coeffVec[4] + coeffVec[5] * t + coeffVec[6] * t2 + coeffVec[7] * t3
                + coeffVec[8] * 2.0 * h + coeffVec[9] * 2.0 * h * t + coeffVec[10] * 2.0 * h * t2 + coeffVec[11] * 2.0 * h * t3
                + coeffVec[12] * 3.0 * h2 + coeffVec[13] * 3.0 * h2 * t + coeffVec[14] * 3.0 * h2 * t2 + coeffVec[15] * 3.0 * h2 * t3;
        }
    }
}

//--------------------------------------------------------------------------------------------------
template<typename _Tp> __global__ void d_interpolBiLi(const float* __restrict__ pts, int numPts,
    const _Tp* __restrict__ imgIn, int width, int height, int step, int colWise, float* __restrict__ intensVals)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int ptId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

    if (ptId < numPts)
    {
        float fIdxPos;
        float sIdxPos;
        int sIdx1, sIdx2, fIdx1, fIdx2;
        int fSize, sSize;
        float DY1, DY2, DX1, DX2;

        if (colWise)
        {
            fIdxPos = pts[ptId * 2 + 1];
            sIdxPos = pts[ptId * 2];
            fSize = height;
            sSize = width;
        }
        else
        {
            fIdxPos = pts[ptId * 2];
            sIdxPos = pts[ptId * 2 + 1];
            fSize = width;
            sSize = height;
        }

        if ((fIdxPos >= fSize) || (sIdxPos >= sSize) || (fIdxPos < 0) || (sIdxPos < 0)
            || !isfinite(fIdxPos) || !isfinite(sIdxPos))
            return;

        if (sIdxPos < (sSize - 1))
        {
            sIdx1 = (int)floor(sIdxPos);
            sIdx2 = sIdx1 + 1;
        }
        else
        {
            sIdx1 = (int)floor(sIdxPos - 1);
            sIdx2 = sIdx1 + 1;
        }

        if (fIdxPos < (fSize - 1))
        {
            fIdx1 = (int)floor(fIdxPos);
            fIdx2 = fIdx1 + 1;
        }
        else
        {
            fIdx1 = (int)floor(fIdxPos - 1);
            fIdx2 = fIdx1 + 1;
        }

        float X1Y1 = imgIn[sIdx1 * step + fIdx1];
        float X1Y2 = imgIn[sIdx2 * step + fIdx1];
        float X2Y1 = imgIn[sIdx1 * step + fIdx2];
        float X2Y2 = imgIn[sIdx2 * step + fIdx2];

        if (!isfinite(X1Y1) || !isfinite(X1Y2) || !isfinite(X2Y1) || !isfinite(X2Y2))
        {
            //intensVals[ptId] = NPP_MAXABS_32F;
            //outPtr[npts] = std::numeric_limits<_Tp>::max();
            return;
        }

        if (colWise)
        {
            DY1 = fIdxPos - fIdx1;
            DY2 = fIdx2 - fIdxPos;
            DX1 = sIdxPos - sIdx1;
            DX2 = sIdx2 - sIdxPos;
        }
        else
        {
            DX1 = fIdxPos - fIdx1;
            DX2 = fIdx2 - fIdxPos;
            DY1 = sIdxPos - sIdx1;
            DY2 = sIdx2 - sIdxPos;
        }

        intensVals[ptId * 3] = (X1Y1 * DX2 * DY2 + X2Y1 * DX1 * DY2 + X1Y2 * DX2 * DY1 + X2Y2 * DX1 * DY1);
        intensVals[ptId * 3 + 1] = (X2Y1 - X1Y1) + ((X2Y2 + X1Y1) - (X2Y1 + X1Y2)) * DY1;
        intensVals[ptId * 3 + 2] = (X1Y2 - X1Y1) + ((X2Y2 + X1Y1) - (X2Y1 + X1Y2)) * DX1;
    }
}

//--------------------------------------------------------------------------------------------------
template<typename _Tp> ito::RetVal h_interpolBiLi(const _Tp *inPtr, const int sizex, const int sizey, const int stepin,
    const float *positions, const int numPos, float *outPtr, const int stepOut, const int flag)
{
    ito::RetVal retval;

    hipError_t cerror;
    dim3 dimGrid;
    dim3 dimBlocks;
    static _Tp *dp_img = NULL;
    static const _Tp *hp_img = NULL;
    static int imgSizex = 0, imgSizey = 0;
    float *dp_pts = NULL, *dp_int = NULL;

    if (numPos <= 256)
    {
        int bs = ceil(sqrt(numPos));
        dimBlocks = dim3(bs, bs);
    }
    else
    {
        dimBlocks = dim3(16, 16);
        int ng1 = ceil(sqrt(numPos / 256.0));
        int ng2 = ceil(numPos / (256.0 * ng1));
        dimGrid = dim3(ng1, ng2);
    }

    // maybe we should refine the combination of checks here. In fact unregistering host memory and right afterwards
    // reregistering it does not work. Anyways this actually should not occur, as, when memory size changed the pointer
    // address should change. So there should be no need to do that.
    if ((hp_img == NULL || dp_img == NULL || imgSizex != sizex || imgSizey != sizey || flag & 4) && (hp_img != inPtr))
    {
        if (hp_img != NULL)
        {
            hipHostUnregister((void*)hp_img);
        }
        hipHostRegister((void*)inPtr, sizex * sizey * sizeof(_Tp), hipHostRegisterMapped);
        hipHostGetDevicePointer(&dp_img, (void*)inPtr, 0);
        if (dp_img == NULL)
        {
            cerror = hipGetLastError();
            return ito::RetVal(ito::retError, 0, hipGetErrorString(cerror));
        }

        hp_img = inPtr;
        imgSizex = sizex;
        imgSizey = sizey;
    }

    hipHostRegister((void*)positions, numPos * 2 * sizeof(float), hipHostRegisterMapped);
    hipHostGetDevicePointer(&dp_pts, (void*)positions, 0);
    hipMalloc((void**)&dp_int, 3 * sizeof(float) * numPos);

    if (flag & 512)
        d_interpolBiLi<_Tp> << <dimGrid, dimBlocks >> >(dp_pts, numPos, dp_img, sizex, sizey, sizey, 1, dp_int);
    else
        d_interpolBiLi<_Tp> << <dimGrid, dimBlocks >> >(dp_pts, numPos, dp_img, sizex, sizey, sizex, 0, dp_int);
    hipDeviceSynchronize();

    hipMemcpy(outPtr, dp_int, 3 * sizeof(float) * numPos, hipMemcpyDeviceToHost);
    if ((cerror = hipGetLastError()))
        retval += ito::RetVal(ito::retError, 0, hipGetErrorString(cerror));

    hipFree(dp_int);
    hipHostUnregister((void*)positions);
    if (flag & 2 != 2)
    {
        hipHostUnregister((void*)hp_img);
        hp_img = NULL;
        dp_img = NULL;
        imgSizex = 0;
        imgSizey = 0;
    }
    if ((cerror = hipGetLastError()))
        return ito::RetVal(ito::retError, 0, hipGetErrorString(cerror));

    return retval;
}

//--------------------------------------------------------------------------------------------------
ito::RetVal h_interpolAMat(const float *inPtr, const int sizex, const int sizey, const int stepin,
    const float *positions, const int numPos, float *outPtr, const int interpAlgo, const int flag)
{
    ito::RetVal retval;

    hipError_t cerror;
    dim3 dimGrid;
    dim3 dimBlocks;
    static const float *dp_img = NULL, *dp_dx = NULL, *dp_dy = NULL, *dp_dxy = NULL;
    static const float *hp_img = NULL, *hp_dx = NULL, *hp_dy = NULL, *hp_dxy = NULL;
    static cv::Mat matDx, matDy, matDxy;
    static int imgSizex = 0, imgSizey = 0;
    const float *dp_pts = NULL;
    float *dp_int = NULL;

    if (numPos <= 256)
    {
        int bs = ceil(sqrt(numPos));
        dimBlocks = dim3(bs, bs);
    }
    else
    {
        dimBlocks = dim3(16, 16);
        int ng1 = ceil(sqrt(numPos / 256.0));
        int ng2 = ceil(numPos / (256.0 * ng1));
        dimGrid = dim3(ng1, ng2);
    }

    if (hp_img == NULL || dp_img == NULL || imgSizex != sizex || imgSizey != sizey || (flag & 4) == 4 || (flag & 2 != 2))
    {
        if (hp_img != NULL)
            hipHostUnregister((void*)hp_img);
        if (hp_dx != NULL)
            hipHostUnregister((void*)hp_dx);
        if (hp_dy != NULL)
            hipHostUnregister((void*)hp_dy);
        if (hp_dxy != NULL)
            hipHostUnregister((void*)hp_dxy);

        if (inPtr == NULL)
            return ito::RetVal(ito::retError, 0, "Error input image pointer is NULL in CUDA interpolate BiCu");
        hipHostRegister((void*)inPtr, sizex * sizey * sizeof(float), hipHostRegisterMapped);
        hipHostGetDevicePointer(&dp_img, (void*)inPtr, 0);

        ito::float32 kernel[9] = { -3.0, 0.0,  3.0,
            -10.0, 0.0, 10.0,
            -3.0, 0.0,  3.0 };
        cv::Mat kernelDx = cv::Mat(cv::Size(3, 3), CV_32F, kernel);
        cv::Mat kernelDy = kernelDx.t();

        //ito::float32 kernel[3] = { -0.5, 0.0, 0.5 };
        //cv::Mat kernelDx = cv::Mat(cv::Size(3, 1), CV_32F, kernel);
        //cv::Mat kernelDy = cv::Mat(cv::Size(1, 3), CV_32F, kernel);

        cv::Mat imgMat(sizey, sizex, CV_32F, (void*)inPtr);

        cv::filter2D(imgMat, matDx, -1, kernelDx, cv::Point(-1, -1), 0.0, cv::BORDER_ISOLATED);
        cv::filter2D(imgMat, matDy, -1, kernelDy, cv::Point(-1, -1), 0.0, cv::BORDER_ISOLATED);
        ito::float32 *dxp = (ito::float32*)matDx.data;
        ito::float32 *dyp = (ito::float32*)matDy.data;

        // border correction
        for (int y = 0; y < sizey; y++)
        {
            dxp[y * sizex] = dxp[y * sizex + 1];
            dxp[y * sizex + sizex - 1] = dxp[y * sizex + sizex - 2];
            dyp[y * sizex] = dyp[y * sizex + 1];
            dyp[y * sizex + sizex - 1] = dyp[y * sizex + sizex - 2];
        }
        for (int x = 0; x < sizex; x++)
        {
            dxp[x] = dxp[sizex + x];
            dxp[(sizey - 1) * sizex + x] = dxp[(sizey - 2) * sizex + x];
            dyp[x] = dyp[sizex + x];
            dyp[(sizey - 1) * sizex + x] = dyp[(sizey - 2) * sizex + x];
        }

        cv::filter2D(matDx, matDxy, -1, kernelDy, cv::Point(-1, -1), 0.0, cv::BORDER_ISOLATED);
        // border correction
        ito::float32 *dxyp = (ito::float32*)matDxy.data;
        for (int x = 0; x < sizex; x++)
        {
            dxyp[x] = dxyp[sizex + x];
            dxyp[(sizey - 1) * sizex + x] = dxyp[(sizey - 2) * sizex + x];
        }
        for (int y = 0; y < sizey; y++)
        {
            dxyp[y * sizex] = dxyp[y * sizex + 1];
            dxyp[y * sizex + sizex - 1] = dxyp[y * sizex + sizex - 2];
        }

        hipHostRegister((void*)dxp, sizex * sizey * sizeof(float), hipHostRegisterMapped);
        hipHostGetDevicePointer(&dp_dx, (void*)dxp, 0);
        hipHostRegister((void*)dyp, sizex * sizey * sizeof(float), hipHostRegisterMapped);
        hipHostGetDevicePointer(&dp_dy, (void*)dyp, 0);
        hipHostRegister((void*)dxyp, sizex * sizey * sizeof(float), hipHostRegisterMapped);
        hipHostGetDevicePointer(&dp_dxy, (void*)dxyp, 0);

        if (dp_img == NULL || dp_dx == NULL || dp_dy == NULL || dp_dxy == NULL)
        {
            cerror = hipGetLastError();
            return ito::RetVal(ito::retError, 0, hipGetErrorString(cerror));
        }
        hp_img = inPtr;
        hp_dx = dxp;
        hp_dy = dyp;
        hp_dxy = dxyp;
        imgSizex = sizex;
        imgSizey = sizey;
    }

    hipHostRegister((void*)positions, numPos * 2 * sizeof(float), hipHostRegisterMapped);
    hipHostGetDevicePointer(&dp_pts, (void*)positions, 0);
    hipMalloc((void**)&dp_int, 3 * sizeof(float) * numPos);

    if (interpAlgo == 1)
    {
        d_interpolBiCu << <dimGrid, dimBlocks >> > (dp_pts, numPos, dp_img, sizex, sizey, sizex, 0, dp_dx, dp_dy, dp_dxy, dp_int);
    }
    else if (interpAlgo == 2)
    {
        //d_interpolBiQi<_Tp> << <dimGrid, dimBlocks >> > (dp_ptsPtr, numPos, dp_imgPtr, sizex, sizey, sizex, 0, dp_intPtr);
    }
    else if (interpAlgo == 3)
    {
        //d_interpolBiHe<_Tp> << <dimGrid, dimBlocks >> > (dp_ptsPtr, numPos, dp_imgPtr, sizex, sizey, sizex, 0, dp_intPtr);
    }
    else
    {
        retval += ito::RetVal(ito::retError, 0, "unknown interpolation algorithm, no output generated!");
    }
    hipDeviceSynchronize();

    hipMemcpy(outPtr, dp_int, 3 * sizeof(float) * numPos, hipMemcpyDeviceToHost);
    if ((cerror = hipGetLastError()))
        retval += ito::RetVal(ito::retError, 0, hipGetErrorString(cerror));

    hipFree(dp_int);
    hipHostUnregister((void*)positions);
    if ((flag & 2) != 2)
    {
        hipHostUnregister((void*)hp_img);
        hp_img = NULL;
        dp_img = NULL;
        hipHostUnregister((void*)hp_dx);
        hp_dx = NULL;
        dp_dx = NULL;
        hipHostUnregister((void*)hp_dy);
        hp_dy = NULL;
        dp_dy = NULL;
        hipHostUnregister((void*)hp_dxy);
        hp_dxy = NULL;
        dp_dxy = NULL;
        imgSizex = 0;
        imgSizey = 0;

        matDx = cv::Mat();
        matDy = cv::Mat();
        matDxy = cv::Mat();
    }
    if ((cerror = hipGetLastError()))
        return ito::RetVal(ito::retError, 0, hipGetErrorString(cerror));

    return retval;
}

//--------------------------------------------------------------------------------------------------
ito::RetVal InitCudaDevice(std::vector<int> &devices)
{
    struct hipDeviceProp_t prop;
    int numdev;
    hipDeviceReset();
    hipGetDeviceCount(&numdev);
    if (numdev == 0)
    {
        return -1;
    }

    for (int r = 0; r < numdev; r++)
    {
        hipGetDeviceProperties(&prop, r);
        if (prop.major >= 1)
        {
            devices.push_back(r);
        }

        if (r == numdev - 1)
        {
            return ito::RetVal(ito::retError, 0, "No CUDA capable device found!\nAborting!\n");
        }
    }

    return ito::retOk;
}

//--------------------------------------------------------------------------------------------------
// template instantiation

template ito::RetVal h_interpolBiLi<unsigned char>(const unsigned char *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<signed char>(const signed char *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<unsigned short>(const unsigned short *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<short>(const short *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<unsigned long>(const unsigned long *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<long>(const long *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<float>(const float *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);
template ito::RetVal h_interpolBiLi<double>(const double *inPtr, const int sizex, const int sizey, const int stepin, const float *positions, const int numPos, float *outPtr, const int stepout, const int flag);

//--------------------------------------------------------------------------------------------------
